#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;

int main(void)
{
  hipDeviceProp_t prop;
  int count;
  hipGetDeviceCount(&count);
  for(int i=0;i<count;i++){

    hipGetDeviceProperties(&prop, i);
    cout<<"Device #"<<i<<"---------------"<<endl;
    cout<<"Name: "<<prop.name<<endl;
    cout<<"multiProcessorCount :"<<prop.multiProcessorCount<<endl;
    cout<<"max thread per bloack :"<<prop.maxThreadsPerBlock<<endl;
  }
return 0;
}